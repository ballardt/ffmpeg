#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

extern "C" {

texture<unsigned char, hipTextureType2D, hipReadModeNormalizedFloat> uchar_tex;

__device__ inline void panomorpth_stretch( float& x, float& y )
{
    float len = sqrtf(x*x+y*y);
    x *= sqrtf(len);
    y *= sqrtf(len);
}

__global__ void Reproject_Fisheye_Equirect_uchar(unsigned char *dst,
                                    int dst_width, int dst_height, int dst_pitch,
                                    int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float xi = xo / (float)dst_width;
        float yi = yo / (float)dst_height;

        // we keep Y unchanged
        // we stretch X
        xi = xi * 2.0f - 1.0f; // xi is now between [-1 and 1]
        yi = yi * 2.0f - 1.0f; // yi is now between [-1 and 1]

        // stretch for a perfect half sphere covering 180 degrees
        // float stretch = sqrtf( 1.0f - yi*yi );

        // This seems to be the perfect stretch factor for our lens, implying
        // our lens cuts through unit sphere at a height above the equator that
        // has only 90% of the equator's radius. That is 0.4358, which appears
        // very far above the equator. We are probably not stretching correctly ...
        float stretch = sqrtf( 0.90f - yi*yi );

        xi *= stretch;

        // panomorpth_stretch(xi,yi);

        xi = ( xi + 1.0f ) / 2.0f; // xf now between [ 0 and 1 ]
        yi = ( yi + 1.0f ) / 2.0f; // xf now between [ 0 and 1 ]

        float y = tex2D(uchar_tex, xi, yi);
        dst[yo*dst_pitch+xo] = (unsigned char)(y*255.0f);
    }
}

}
